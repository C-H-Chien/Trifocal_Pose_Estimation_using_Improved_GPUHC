#include "hip/hip_runtime.h"
#ifndef kernel_HC_Solver_trifocal_2op1p_30_direct_param_homotopy_mb_cu
#define kernel_HC_Solver_trifocal_2op1p_30_direct_param_homotopy_mb_cu
// ============================================================================
// GPU homotopy continuation solver for the trifocal 2op1p 30x30 problem
// Version 2: Direct evaluation of parameter homotopy. The parameter homotopy 
//            part of each polynomial is not expanded to an uni-variable 
//            polynomial. Rather, depending on the order of t, the parameter 
//            homotopy formulation is explicitly hard-coded such that we do not 
//            need to compute coefficients from parameters first then use 
//            index-based to evaluate the Jacobians. The required amount of data 
//            to be stored in a kernel in this method is reduced which expects 
//            to speedup over the first version.
//
// Major Modifications
//    Chiang-Heng Chien  22-10-03:   Edited from the first version 
//                                   (kernel_HC_Solver_trifocal_2op1p_30.cu)
//    Chiang-Heng Chien  23-07-18:   Run under a RANSAC scheme with multiple 
//                                   batches and multiple HC trackins per warp
//
// ============================================================================
#include <stdio.h>
#include <stdlib.h>
#include <cstdio>
#include <iostream>
#include <iomanip>
#include <cstring>

// cuda included
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// magma
#include "flops.h"
#include "magma_v2.h"
#include "magma_lapack.h"
#include "magma_internal.h"
#undef max
#undef min
#include "magma_templates.h"
#include "sync.cuh"
#undef max
#undef min
#include "shuffle.cuh"
#undef max
#undef min
#include "batched_kernel_param.h"

//> header
#include "magmaHC-kernels.h"

//> device functions
#include "../gpu-dev-functions/dev-eval-indxing-trifocal_2op1p_30_direct_param_homotopy.cuh"
#include "../gpu-dev-functions/dev-cgesv-batched-small.cuh"
#include "../gpu-dev-functions/dev-get-new-data.cuh"

#include "../../definitions.h"

namespace magmaHCWrapper {

  template<int N, int num_of_params, int max_steps, int max_corr_steps, int predSuccessCount, 
           int Hx_max_terms, int Hx_max_parts, int Hx_max_terms_parts, int Ht_max_terms, int Ht_max_parts,
           int batchCount, int NUMBER_OF_BATCHES_MULTIPLES, int NUMBER_OF_TRACKINGS_PER_WARP>
  __global__ void
  HC_solver_trifocal_2op1p_30_direct_param_homotopy_mb(
    magma_int_t ldda, 
    magmaFloatComplex** d_startSols_array,
    magmaFloatComplex** d_Track_array,
    magmaFloatComplex*  d_startParams,
    magmaFloatComplex*  d_targetParams,
    magmaFloatComplex** d_cgesvA_array,
    magmaFloatComplex** d_cgesvB_array,
    magmaFloatComplex*  d_diffParams,
    const magma_int_t* __restrict__ d_Hx_indices,
    const magma_int_t* __restrict__ d_Ht_indices,
    magmaFloatComplex*  d_path_converge_flag
  )
  {
    extern __shared__ magmaFloatComplex zdata[];
    const int tx = threadIdx.x;
    const int batchid = blockIdx.x ;

    //> define pointers to the arrays
    magmaFloatComplex* d_startSols    = d_startSols_array[batchid];
    magmaFloatComplex* d_cgesvA       = d_cgesvA_array[batchid];
    magmaFloatComplex* d_cgesvB       = d_cgesvB_array[batchid];

    //> declarations of registers
    magmaFloatComplex r_cgesvA[N] = {MAGMA_C_ZERO};
    magmaFloatComplex r_cgesvB    = MAGMA_C_ZERO;
    
    //> declarations of shared memories
    magmaFloatComplex *s_startParams        = (magmaFloatComplex*)(zdata);
    magmaFloatComplex *s_targetParams       = s_startParams + (num_of_params + 1);
    magmaFloatComplex *s_diffParams         = s_targetParams + (num_of_params + 1);
    magmaFloatComplex *s_param_homotopy     = s_diffParams + (num_of_params + 1);
    magmaFloatComplex *s_sols               = s_param_homotopy + (num_of_params + 1);
    magmaFloatComplex *s_track              = s_sols + (N+1);
    magmaFloatComplex *s_track_last_success = s_track + (N+1);
    magmaFloatComplex *sB                   = s_track_last_success + (N+1);
    magmaFloatComplex *sx                   = sB + N;
    float *dsx                              = (float*)(sx + N);
    float *s_sqrt_sols                      = dsx + N;
    float *s_sqrt_corr                      = s_sqrt_sols + N;
    float *s_norm                           = s_sqrt_corr + N;
    int   *sipiv                            = (int*)(s_norm + 2);
    bool   s_isSuccessful                   = (bool)(sipiv + N);
    int    s_pred_success_count             = (int)(s_isSuccessful + 1);

    //> read data from global memory to registers
    #pragma unroll
    for(int i = 0; i < N; i++) {
      r_cgesvA[i] = d_cgesvA[ i * ldda + tx ];
    }
    r_cgesvB = d_cgesvB[tx];

    //> start and target parameters
    s_startParams[tx]  = d_startParams[tx];
    
    if (tx == 0) {
      //> the rest of the start and target parameters
      #pragma unroll
      for(int i = N; i <= num_of_params; i++) {
        s_startParams[i]  = d_startParams[i];
        //s_targetParams[i] = d_targetParams[i];
        //s_diffParams[i]   = d_diffParams[i];
      }
      s_sols[N]                       = MAGMA_C_MAKE(1.0, 0.0);
      s_track[N]                      = MAGMA_C_MAKE(1.0, 0.0);
      s_track_last_success[N]         = MAGMA_C_MAKE(1.0, 0.0);
      s_param_homotopy[num_of_params] = MAGMA_C_ONE;
    }
    __syncthreads();

    //> 1/2 \Delta t
    float one_half_delta_t;

    //>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
    int batch_mul_id = batchid / batchCount;
    for (int ri = 0; ri < NUMBER_OF_TRACKINGS_PER_WARP; ri++) {
      magmaFloatComplex* d_track = d_Track_array[batchid + ri*batchCount*NUMBER_OF_BATCHES_MULTIPLES];
      s_track[tx]                = d_track[tx];

      
      s_targetParams[tx]         = d_targetParams[tx + batch_mul_id * (num_of_params+1) + ri * NUMBER_OF_BATCHES_MULTIPLES * (num_of_params+1)];
      s_diffParams[tx]           = d_diffParams[tx + batch_mul_id * (num_of_params+1) + ri * NUMBER_OF_BATCHES_MULTIPLES * (num_of_params+1)];

      s_sols[tx]               = d_startSols[tx];
      s_track_last_success[tx] = s_track[tx];
      s_sqrt_sols[tx]          = 0;
      s_sqrt_corr[tx]          = 0;
      s_isSuccessful           = 0;
      s_pred_success_count     = 0;
      __syncthreads();

      int linfo = 0, rowid = tx;
      float t0 = 0.0, t_step = 0.0, delta_t = 0.01;
      bool end_zone = 0;
      int hc_step = 0;

      #pragma unroll
      for(int i = N; i <= num_of_params; i++) {
        s_targetParams[i] = d_targetParams[i];
        s_diffParams[i]   = d_diffParams[i];
      }

      //>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
      /*if (tx == 0 && batchid == 1) {
        printf("Round #%d\n", ri);
        for (int ii = 0; ii < 31; ii++) {
          printf("%.5f\t%.5f\n", MAGMA_C_REAL(s_track[ii]), MAGMA_C_IMAG(s_track[ii]));
        }
        printf("\n");
      }

      if (tx == 0 && batchid == 1) {
        printf("Round #%d\n", ri);
        for (int ii = 0; ii < 34; ii++) {
          printf("%.5f\t%.5f\n", MAGMA_C_REAL(s_targetParams[ii]), MAGMA_C_IMAG(s_targetParams[ii]));
        }
        printf("\n");
      }*/

      //#pragma unroll
      for (int step = 0; step <= max_steps; step++) {
        if (t0 < 1.0 && (1.0-t0 > 0.0000001)) {

          // ===================================================================
          // Decide delta t at end zone
          // ===================================================================
          if (!end_zone && fabs(1 - t0) <= (0.0500001)) {
            end_zone = true;

            //> TEST!!!!!!!!!!!!!!!!!!!!!
            //break;
          }

          if (end_zone) {
            if (delta_t > fabs(1 - t0))
              delta_t = fabs(1 - t0);
          }
          else if (delta_t > fabs(0.95 - t0)) {
            delta_t = fabs(0.95 - t0);
          }

          t_step = t0;
          one_half_delta_t = 0.5 * delta_t;
          // ===================================================================
          // Prediction: 4-th order Runge-Kutta method
          // ===================================================================
          //>  get HxHt for k1
          compute_param_homotopy<N>( tx, t0, s_param_homotopy, s_startParams, s_targetParams );
          eval_Jacobian_Hx<N, Hx_max_terms, Hx_max_parts, Hx_max_terms_parts>( tx, t0, r_cgesvA, s_track, s_startParams, s_targetParams, s_param_homotopy, d_Hx_indices );
          eval_Jacobian_Ht<N, Ht_max_terms, Ht_max_parts>( tx, t0, r_cgesvB, s_track, s_startParams, s_targetParams, s_param_homotopy, d_Ht_indices, s_diffParams );

          // -- solve k1 --
          cgesv_batched_small_device<N>( tx, r_cgesvA, sipiv, r_cgesvB, sB, sx, dsx, rowid, linfo );
          magmablas_syncwarp();

          // -- compute x for the creation of HxHt for k2 --
          create_x_for_k2( tx, t0, delta_t, one_half_delta_t, s_sols, s_track, sB );
          magmablas_syncwarp();

          // -- get HxHt for k2 --
          compute_param_homotopy<N>( tx, t0, s_param_homotopy, s_startParams, s_targetParams );
          eval_Jacobian_Hx<N, Hx_max_terms, Hx_max_parts, Hx_max_terms_parts>( tx, t0, r_cgesvA, s_track, s_startParams, s_targetParams, s_param_homotopy, d_Hx_indices );
          eval_Jacobian_Ht<N, Ht_max_terms, Ht_max_parts>( tx, t0, r_cgesvB, s_track, s_startParams, s_targetParams, s_param_homotopy, d_Ht_indices, s_diffParams );

          // -- solve k2 --
          cgesv_batched_small_device<N>( tx, r_cgesvA, sipiv, r_cgesvB, sB, sx, dsx, rowid, linfo );
          magmablas_syncwarp();

          // -- compute x for the generation of HxHt for k3 --
          create_x_for_k3( tx, delta_t, one_half_delta_t, s_sols, s_track, s_track_last_success, sB );
          magmablas_syncwarp();

          // -- get HxHt for k3 --
          //compute_param_homotopy<N>( tx, t0, s_param_homotopy, s_start_params, s_target_params );
          eval_Jacobian_Hx<N, Hx_max_terms, Hx_max_parts, Hx_max_terms_parts>( tx, t0, r_cgesvA, s_track, s_startParams, s_targetParams, s_param_homotopy, d_Hx_indices );
          eval_Jacobian_Ht<N, Ht_max_terms, Ht_max_parts>( tx, t0, r_cgesvB, s_track, s_startParams, s_targetParams, s_param_homotopy, d_Ht_indices, s_diffParams );

          // -- solve k3 --
          cgesv_batched_small_device<N>( tx, r_cgesvA, sipiv, r_cgesvB, sB, sx, dsx, rowid, linfo );
          magmablas_syncwarp();

          // -- compute x for the generation of HxHt for k4 --
          create_x_for_k4( tx, t0, delta_t, one_half_delta_t, s_sols, s_track, s_track_last_success, sB );
          magmablas_syncwarp();

          // -- get HxHt for k4 --
          compute_param_homotopy<N>( tx, t0, s_param_homotopy, s_startParams, s_targetParams );
          eval_Jacobian_Hx<N, Hx_max_terms, Hx_max_parts, Hx_max_terms_parts>( tx, t0, r_cgesvA, s_track, s_startParams, s_targetParams, s_param_homotopy, d_Hx_indices );
          eval_Jacobian_Ht<N, Ht_max_terms, Ht_max_parts>( tx, t0, r_cgesvB, s_track, s_startParams, s_targetParams, s_param_homotopy, d_Ht_indices, s_diffParams );

          // -- solve k4 --
          cgesv_batched_small_device<N>( tx, r_cgesvA, sipiv, r_cgesvB, sB, sx, dsx, rowid, linfo );
          magmablas_syncwarp();

          // -- make prediction --
          s_sols[tx] += sB[tx] * delta_t * 1.0/6.0;
          s_track[tx] = s_sols[tx];
          __syncthreads();

          // ===================================================================
          // -- Gauss-Newton Corrector --
          // ===================================================================
          //#pragma unroll
          for(int i = 0; i < max_corr_steps; i++) {

            //> evaluate the Jacobian Hx
            eval_Jacobian_Hx<N, Hx_max_terms, Hx_max_parts, Hx_max_terms_parts>( tx, t0, r_cgesvA, s_track, s_startParams, s_targetParams, s_param_homotopy, d_Hx_indices );

            //> evaluate the parameter homotopy
            eval_Parameter_Homotopy<N, Ht_max_terms, Ht_max_parts>( tx, t0, r_cgesvB, s_track, s_startParams, s_targetParams, s_param_homotopy, d_Ht_indices );

            //> G-N corrector first solve
            cgesv_batched_small_device<N>( tx, r_cgesvA, sipiv, r_cgesvB, sB, sx, dsx, rowid, linfo );
            magmablas_syncwarp();

            //> correct the sols
            s_track[tx] -= sB[tx];
            __syncthreads();

            //> compute the norms; norm[0] is norm(sB), norm[1] is norm(sol)
            compute_norm2<N>( tx, sB, s_track, s_sqrt_sols, s_sqrt_corr, s_norm );
            __syncthreads();
            
            s_isSuccessful = s_norm[0] < 0.000001 * s_norm[1];
            __syncthreads();

            if (s_isSuccessful)
              break;
          }

          //> stop if the values of the solution is too large
          if ((s_norm[1] > 1e14) && (t0 < 1.0) && (1.0-t0 > 0.001)) {
            //inf_failed = 1;
            break;
          }

          // ===================================================================
          // Decide Track Changes
          // ===================================================================
          if (!s_isSuccessful) {
            s_pred_success_count = 0;
            delta_t *= 0.5;
            //> should be the last successful tracked sols
            s_track[tx] = s_track_last_success[tx];
            s_sols[tx] = s_track_last_success[tx];
            __syncthreads();
            t0 = t_step;
          }
          else {
            s_track_last_success[tx] = s_track[tx];
            s_sols[tx] = s_track[tx];
            __syncthreads();
            s_pred_success_count++;
            if (s_pred_success_count >= predSuccessCount) {
              s_pred_success_count = 0;
              delta_t *= 2;
            }
          }
          hc_step++;
        }
        else {
          break;
        }
      }

      //> d_cgesvB tells whether the track is finished, if not, stores t0 and delta_t
      d_path_converge_flag[batchid + ri*batchCount] = (t0 >= 1.0 || (1.0-t0 <= 0.0000001)) ? MAGMA_C_MAKE(1.0, hc_step) : MAGMA_C_MAKE(t0, delta_t);

      //> d_track stores the solutions
      d_track[tx] = s_track[tx];
    }
  }

  extern "C" real_Double_t
  kernel_HC_Solver_trifocal_2op1p_30_direct_param_homotopy_mb(
    magma_queue_t my_queue,
    magma_int_t ldda,
    magma_int_t N, 
    magma_int_t num_of_params,
    magma_int_t batchCount, 
    magmaFloatComplex** d_startSols_array, 
    magmaFloatComplex** d_Track_array, //
    magmaFloatComplex*  d_startParams,
    magmaFloatComplex*  d_targetParams, //
    magmaFloatComplex** d_cgesvA_array, 
    magmaFloatComplex** d_cgesvB_array,
    magmaFloatComplex*  d_diffParams, //
    magma_int_t* d_Hx_indx, 
    magma_int_t* d_Ht_indx,
    magmaFloatComplex*  d_path_converge_flag)
  {
    real_Double_t gpu_time;
    const magma_int_t thread_x = N;
    dim3 threads(thread_x, 1, 1);
    dim3 grid(batchCount, 1, 1);
    hipError_t e = hipErrorInvalidValue;

    //std::cout << "batchCount = " << batchCount << std::endl;

    //> declare the amount of shared memory for the use of the kernel
    magma_int_t shmem  = 0;
    shmem += (num_of_params+1) * sizeof(magmaFloatComplex);  //> start parameters
    shmem += (num_of_params+1) * sizeof(magmaFloatComplex);  //> target parameters
    shmem += (num_of_params+1) * sizeof(magmaFloatComplex);  //> difference of start and target parameters
    shmem += (num_of_params+1) * sizeof(magmaFloatComplex);  //> parameter homotopy used when t is changed
    shmem += (N+1) * sizeof(magmaFloatComplex);              //> start solutions
    shmem += (N+1) * sizeof(magmaFloatComplex);              //> intermediate solutions
    shmem += (N+1) * sizeof(magmaFloatComplex);              //> last successful intermediate solutions
    shmem += N * sizeof(magmaFloatComplex);                  //> linear system solution
    shmem += N * sizeof(magmaFloatComplex);                  //> intermediate varaible for cgesv
    shmem += N * sizeof(float);                              //> intermediate varaible for cgesv
    shmem += N * sizeof(int);                                //> squared solution
    shmem += N * sizeof(float);                              //> squared correction solution
    shmem += N * sizeof(float);                              //> solution norm
    shmem += 2 * sizeof(float);                              //> pivot for cgesv
    shmem += 1 * sizeof(bool);                               //> is_successful 
    shmem += 1 * sizeof(int);                                //> predictor successes counter

    //> declare kernel arguments  
    void *kernel_args[] = {&ldda, 
                           &d_startSols_array,
                           &d_Track_array,
                           &d_startParams, &d_targetParams,
                           &d_cgesvA_array, &d_cgesvB_array,
                           &d_diffParams,
                           &d_Hx_indx, &d_Ht_indx,
                           &d_path_converge_flag
                          };

    gpu_time = magma_sync_wtime( my_queue );

    //int batchCount, int NUMBER_OF_BATCHES_MULTIPLES, int NUMBER_OF_TRACKINGS_PER_WARP>

    //> launch the GPU kernel
    //> < Number of Unknowns, Number of Parameters, Maximal Steps, Number of correction steps, Number of steps to be successful, Don't care...>
    //> LAST THREE ARGUMENTS: (int batchCount, int NUMBER_OF_BATCHES_MULTIPLES, int NUMBER_OF_TRACKINGS_PER_WARP)
    e = hipLaunchKernel((void*)HC_solver_trifocal_2op1p_30_direct_param_homotopy_mb
                         < 30, 33, 100, 5, 10, 8, 5, 40, 16, 6, 312, MULTIPLES_OF_BATCHCOUNT, MULTIPLES_OF_TRACKING_PER_WARP >, 
                         grid, threads, kernel_args, shmem, my_queue->hip_stream());


    gpu_time = magma_sync_wtime( my_queue ) - gpu_time;
    if( e != hipSuccess ) {
        printf("hipLaunchKernel of HC_solver_trifocal_2op1p_30_direct_param_homotopy is not successful!\n");
    }

    return gpu_time;
  }

}

#endif
